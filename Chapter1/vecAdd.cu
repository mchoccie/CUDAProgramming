
#include <hip/hip_runtime.h>
void vecAdd(float * A_h, float * B_h, float * c_h, int n){
    int size = n * sizeof(float);
    float * A_d, * B_d, * c_d;
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&c_d, size);
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
}