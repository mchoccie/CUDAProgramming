#include "hip/hip_runtime.h"
__global___ void vecAddKernel(float * A, float * B, float * c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = A[i] + B[i];
    }
}
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <chrono>

void vecAdd(float * A_h, float * B_h, float * c_h, int n){
    int size = n * sizeof(float);
    float * A_d, * B_d, * c_d;
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&c_d, size);
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, c_d, n);
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(c_d);
}